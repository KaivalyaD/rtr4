#include "hip/hip_runtime.h"
/* header files */
// standard headers
#include<stdio.h>

// CUDA headers
#include<hip/hip_runtime.h>
#include"include/helper_timer.h"

/* macros */
#define BLOCK_WIDTH 1024

/* global variable declarations */
int *hostA = NULL;
int *hostB = NULL;
int *hostC = NULL;
int *gold = NULL;

int *deviceA = NULL;
int *deviceB = NULL;
int *deviceC = NULL;

/* CUDA kernel */
__global__
void
multiplyMatrices_GPU (
	const int *const A,
	const int *const B,
	int *const C,
	const int numRowsA,
	const int numColsA,
	const int numColsB,
	const int numColsC
) {
	// variable declarations
	int row, column;

	// code
	row = blockIdx.y * blockDim.y + threadIdx.y;
	column = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < numRowsA && column < numColsB)
	{
		int e, f, acc = 0;
		for (int k = 0; k < numColsA; k++)
		{
			e = *(A + (row * numColsA) + k);	 // (row * numColsA) is a constant for each SP that freezes a row
			f = *(B + column + (k * numColsB)); // column is also a constant for that SP which freezes a column
			
			/*
			 * multiply corresponding elements in the fixed row-column pair
			 * and accumulate their sum (an O(n) operation)
			 * Each row-column pair is therefore processed parallely
			 */
			acc += (e * f);
		}

		// finally, assign the accumulate to its appropriate position in the resultant matrix
		*(C + (row * numColsC) + column) = acc;
	}
}

/* entry-point function */
int main(void)
{
	// function prototypes
	void initA(int *const, const int, const int);
	void initB(int *const, const int, const int);
	void multiplyMatrices_CPU(
		const int *const,
		const int *const,
		int *const,
		const int,
		const int,
		const int,
		const int
	);
	void cleanup(void);

	// variable declarations
	int numRowsA = BLOCK_WIDTH;
	int numColsA = BLOCK_WIDTH;
	int numRowsB = BLOCK_WIDTH;
	int numColsB = BLOCK_WIDTH;
	
	int numRowsC = numRowsA;
	int numColsC = numColsB;
	int numRowsGold = numRowsC;
	int numColsGold = numColsC;

	size_t sizeA = numRowsA * numColsA * sizeof(int);
	size_t sizeB = numRowsB * numColsB * sizeof(int);
	size_t sizeC = numRowsC * numColsC * sizeof(int);
	size_t sizeGold = numRowsGold * numColsGold * sizeof(int);

	float timeOnGPU, timeOnCPU;

	hipError_t result = hipSuccess;

	// code
	// host memory allocation
	hostA = (int *)malloc(sizeA);
	if (!hostA)
	{
		printf("malloc: failed to allocate memory for host buffer hostA\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	hostB = (int *)malloc(sizeB);
	if (!hostB)
	{
		printf("malloc: failed to allocate memory for host buffer hostB\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	hostC = (int *)malloc(sizeC);
	if (!hostC)
	{
		printf("malloc: failed to allocate memory for host buffer hostC\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	gold = (int *)malloc(sizeGold);
	if (!gold)
	{
		printf("malloc: failed to allocate memory for host buffer gold\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	// display pre-operations information
	printf("\n");

	printf("Dimensions of Matrices:\n");
	printf("\tA:\t%d x %d\n", numRowsA, numColsA);
	printf("\tB:\t%d x %d\n", numRowsB, numColsB);
	printf("\tC:\t%d x %d\n", numRowsC, numColsC);
	printf("\tgold:\t%d x %d\n\n", numRowsGold, numColsGold);

	printf("Sizes (in bytes) of Matrices:\n");
	printf("\tA:\t%llu\n", sizeA);
	printf("\tB:\t%llu\n", sizeB);
	printf("\tC:\t%llu\n", sizeC);
	printf("\tgold:\t%llu\n\n", sizeGold);

	// populate host input buffers
	initA(hostA, numRowsA, numColsA);
	initB(hostB, numRowsB, numColsB);

	// device memory allocation
	result = hipMalloc((void **)&deviceA, sizeA);
	if (result != hipSuccess)
	{
		printf("hipMalloc: failed to allocate memory for device buffer deviceA\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	result = hipMalloc((void **)&deviceB, sizeB);
	if (result != hipSuccess)
	{
		printf("hipMalloc: failed to allocate memory for device buffer deviceB\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	result = hipMalloc((void **)&deviceC, sizeC);
	if (result != hipSuccess)
	{
		printf("hipMalloc: failed to allocate memory for device buffer deviceC\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	// populate device input buffers
	result = hipMemcpy(deviceA, hostA, sizeA, hipMemcpyHostToDevice);
	if (result != hipSuccess)
	{
		printf("hipMemcpy: failed to copy host buffer hostA into device buffer deviceA\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	result = hipMemcpy(deviceB, hostB, sizeB, hipMemcpyHostToDevice);
	if (result != hipSuccess)
	{
		printf("hipMemcpy: failed to copy host buffer hostB into device buffer deviceB\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	// configure the kernel
	dim3 dimGrid = dim3(ceil((int)numColsB / (int)BLOCK_WIDTH), ceil((int)numRowsA / (int)BLOCK_WIDTH));
	dim3 dimBlock = dim3(BLOCK_WIDTH, BLOCK_WIDTH, 1);

	// configure the timer
	StopWatchInterface *timer = NULL;
	sdkCreateTimer(&timer);
	
	// start the timer
	sdkStartTimer(&timer);
	{
		// launch the kernel
		multiplyMatrices_GPU<<< dimGrid, dimBlock >>>(
			deviceA,
			deviceB,
			deviceC,
			numRowsA,
			numColsA,
			numColsB,
			numColsGold
		);
	}
	// stop the timer
	sdkStopTimer(&timer);

	// extract time spent on GPU
	timeOnGPU = sdkGetTimerValue(&timer);
	
	// delete the timer
	sdkDeleteTimer(&timer);
	timer = NULL;

	// copy resultant matrix from device output buffer to host output buffer
	result = hipMemcpy(hostC, deviceC, sizeC, hipMemcpyDeviceToHost);
	if (result != hipSuccess)
	{
		printf("hipMemcpy: failed to copy device buffer deviceC into host buffer hostC\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	// reconfigure timer
	sdkCreateTimer(&timer);

	// restart the timer
	sdkStartTimer(&timer);
	{
		// operate on the CPU
		multiplyMatrices_CPU(
			hostA,
			hostB,
			gold,
			numRowsA,
			numColsA,
			numColsB,
			numColsC
		);
	}
	// stop the timer
	sdkStopTimer(&timer);

	// extract time spent on CPU
	timeOnCPU = sdkGetTimerValue(&timer);

	// delete the timer
	sdkDeleteTimer(&timer);
	timer = NULL;

	// compare accuracies
	const float epsilon = 0.000001f;
	bool isAccurate = true;
	int firstInaccurateIndex = -1;
	char statementOfAccuracy[128];

	for (int i = 0; i < numRowsC * numColsC; i++)
	{
		if (fabs(hostC[i] - gold[i]) > epsilon)
		{
			isAccurate = false;
			firstInaccurateIndex = i;
			break;
		}
	}
	if (!isAccurate)
	{
		sprintf(
			statementOfAccuracy,
			"The GPU produced atleast 1 inaccurate result at %d such that its deviation from the actual result is >%.6f",
			firstInaccurateIndex,
			epsilon);
	}
	else
	{
		sprintf(
			statementOfAccuracy,
			"All GPU results are accurate atleast up to %.6f",
			epsilon
		);
	}

	// compare performance
	float performanceRatio = timeOnCPU / timeOnGPU;
	char statementOfPerformance[128];

	if (performanceRatio < 1.0f)
		sprintf(statementOfPerformance, "CPU outperformed the GPU by a factor of %.6f", 1 / performanceRatio);
	else
		sprintf(statementOfPerformance, "GPU outperformed the CPU by a factor of %.6f", performanceRatio);

	// display results of the benchmark
	printf("Results:\n\n");

	printf("\tTime spent on the CPU for Matrix Multiplication A * B = C: %.6f ms\n\n", timeOnCPU);
	
	printf("\tTime spent on the GPU for Matrix Multiplication A * B = C: %.6f ms\n\n", timeOnGPU);
	
	printf("CPU vs GPU with respect to Accuracy:\n");
	printf("\t%s\n", statementOfAccuracy);

	printf("CPU vs GPU with respect to performance:\n");
	printf("\t%s\n", statementOfPerformance);

	// cleanup
	cleanup();

	return 0;
}

void initA(int *const buffer, const int countRows, const int countCols)
{
	// variable declarations
	int v = 1;

	// code
	for (int i = 0; i < countRows; i++)
	{
		for (int j = 0; j < countCols; j++)
		{
			*(buffer + (i * countCols) + j) = v;
			v++;
		}
	}
}

void initB(int *const buffer, const int countRows, const int countCols)
{
	// variable declarations
	int v = BLOCK_WIDTH;

	// code
	for (int i = 0; i < countRows; i++)
	{
		for (int j = 0; j < countCols; j++)
		{
			*(buffer + (i * countCols) + j) = v;
			v--;
		}
	}
}

void
multiplyMatrices_CPU(
	const int *const A,
	const int *const B,
	int *const C,
	const int numRowsA,
	const int numColsA,
	const int numColsB,
	const int numColsC
) {
	// variable declarations
	int row, column;

	for (row = 0; row < numRowsA; row++)
	{
		for (column = 0; column < numColsB; column++)
		{
			int e1, e2, acc = 0;

			for (int k = 0; k < numColsA; k++)
			{
				e1 = *(A + (row * numColsA) + k);
				e2 = *(B + column + (k * numColsB));

				acc += e1 * e2;
			}

			*(C + (row * numColsC) + column) = acc;
		}
	}
}

void cleanup(void)
{
	// code
	if (deviceC)
	{
		hipFree(deviceC);
		deviceC = NULL;
	}
	if (deviceB)
	{
		hipFree(deviceB);
		deviceB = NULL;
	}
	if (deviceA)
	{
		hipFree(deviceA);
		deviceA = NULL;
	}
	if (gold)
	{
		free(gold);
		gold = NULL;
	}
	if (hostC)
	{
		free(hostC);
		hostC = NULL;
	}
	if (hostB)
	{
		free(hostB);
		hostB = NULL;
	}
	if (hostA)
	{
		free(hostA);
		hostA = NULL;
	}
}
