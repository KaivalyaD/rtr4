/* header files */
// standard headers
#include<stdio.h>

// CUDA headers
#include<hip/hip_runtime.h>

/* entry-point function */
int main(void)
{
	// function prototypes
	void showCudaDeviceProperties(void);

	// code
	showCudaDeviceProperties();

	return 0;
}

void showCudaDeviceProperties(void)
{
	// variable declarations
	hipError_t retCudaRT;
	int devCount;

	// code
	printf("\n================================================== CUDA INFORMATION ===============================================\n\n");
	
	retCudaRT = hipGetDeviceCount(&devCount);
	if (retCudaRT != hipSuccess)
	{
		printf("in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(retCudaRT));
		exit(EXIT_FAILURE);
	}
	else if (devCount == 0)
	{
		printf("There is no CUDA-supported device on this system\n");
		return;
	}
	else
	{
		printf("Total number of CUDA-supported devices on this system : %d\n", devCount);

		for (int i = 0; i < devCount; i++)
		{
			hipDeviceProp_t devProp;
			int driverVersion = 0, runtimeVersion = 0;

			retCudaRT = hipGetDeviceProperties(&devProp, i);
			if (retCudaRT != hipSuccess)
			{
				printf("in %s at line %d : %s\n", __FILE__, __LINE__, hipGetErrorString(retCudaRT));
				exit(EXIT_FAILURE);
			}

			hipDriverGetVersion(&driverVersion);
			hipRuntimeGetVersion(&runtimeVersion);
			printf("\n======================================= CUDA DRIVER AND RUNTIME INFORMATION =======================================\n\n");

			printf("CUDA Driver Version  : %d.%d\n", driverVersion / 1000, (driverVersion % 100) / 10);
			printf("CUDA Runtime Version : %d.%d\n", runtimeVersion / 1000, (runtimeVersion % 100) / 10);
			
			printf("\n========================================== GPU DEVICE GENERAL INFORMATION =========================================\n\n");

			printf("GPU Device Number             : %d\n", i);
			printf("GPU Device Name               : %s\n", devProp.name);
			printf("GPU Device Compute Capability : %d.%d\n", devProp.major, devProp.minor);
			printf("GPU Device Clock Rate         : %lu\n", (unsigned long)devProp.clockRate);
			printf("GPU Device Type               : ");
			if (devProp.integrated)
				printf("Integrated (On-Board)\n");
			else
				printf("Discrete\n");

			printf("\n=========================================== GPU DEVICE MEMORY INFORMATION =========================================\n\n");

			printf("Total Memory                  : %.0fGiB = %.0fMiB = %llu Bytes\n",
				((float)devProp.totalGlobalMem / (1024.0f * 1024.0f * 1024.0f)),
				((float)devProp.totalGlobalMem / (1024.0f * 1024.0f)),
				(unsigned long long)devProp.totalGlobalMem);
			printf("Constant Memory               : %lu Bytes\n", (unsigned long)devProp.totalConstMem);
			printf("Shared Memory Per SMProcessor : %lu Bytes\n", (unsigned long)devProp.sharedMemPerBlock);

			printf("\n======================================= GPU DEVICE MULTIPROCESSOR INFORMATION =====================================\n\n");

			printf("Number of SMProcessors              : %d\n", devProp.multiProcessorCount);
			printf("Number of Registers per SMProcessor : %d\n", devProp.regsPerBlock);
			
			printf("\n=========================================== GPU DEVICE THREAD INFORMATION =========================================\n\n");

			printf("Max. number of Threads per SMProcessor : %d\n", devProp.maxThreadsPerMultiProcessor);
			printf("Max. number of Threads per Block       : %d\n", devProp.maxThreadsPerBlock);
			printf("Threads in Warp                        : %d\n", devProp.warpSize);
			printf("Max. Thread Dimensions                 : %d x %d x %d\n", devProp.maxThreadsDim[0], devProp.maxThreadsDim[1], devProp.maxThreadsDim[2]);
			printf("Max. Grid Dimensions                   : %d x %d x %d\n", devProp.maxGridSize[0], devProp.maxGridSize[1], devProp.maxGridSize[2]);

			printf("\n=========================================== GPU DEVICE DRIVER INFORMATION =========================================\n\n");
			
			printf("ECC Support                    : %s\n", devProp.ECCEnabled ? "Enabled" : "Disabled");

#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
			printf("CUDA Driver Mode (TCC or WDDM) : %s\n", devProp.tccDriver ? "TCC (Tesla Compute Cluster)" : "WDDM (Windows Display Driver Model)");
#endif

			printf("\n*******************************************************************************************************************\n\n");
		}
	}
}
