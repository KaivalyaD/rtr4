/* header files */
// standard
#include<stdio.h>	// for printf()

// CUDA
#include<hip/hip_runtime.h>

/* global variables */
const int element_count = 5;

float *host_input_1 = NULL;
float *host_input_2 = NULL;
float *host_output = NULL;

float *device_input_1 = NULL;
float *device_input_2 = NULL;
float *device_output = NULL;

/* CUDA kernel */
__global__ void vectAdd_GPU(float *a, float *b, float *out, int len)
{
	// code
	int i = blockIdx.x * blockDim.x + threadIdx.x;	// i represents ID of this hardware thread

	if (i < len)
	{
		*(out + i) = *(a + i) + *(b + i);
	}
}

/* entry-point function */
int main(void)
{
	// function declarations
	void cleanup(void);

	// variable declarations
	int size = element_count * sizeof(float), i;
	hipError_t result = hipSuccess;

	// code
	// host memory allocation
	host_input_1 = (float *)malloc(size);
	if (!host_input_1)
	{
		printf("malloc: failed to allocate memory for host_input_1\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	host_input_2 = (float *)malloc(size);
	if (!host_input_2)
	{
		printf("malloc: failed to allocate memory on host for host_input_2\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	host_output = (float *)malloc(size);
	if (!host_output)
	{
		printf("malloc: failed to allocate memory on host for host_output\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	// filling the arrays
	for (i = 0; i < element_count; i++)
	{
		host_input_1[i] = 101.0f + i;
		host_input_2[i] = 201.0f + i;
	}

	// device memory allocation
	result = hipMalloc((void **)&device_input_1, size);
	if (result != hipSuccess)
	{
		printf("cudaMalloc: failed to allocate memory on device for device_input_1\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	result = hipMalloc((void **)&device_input_2, size);
	if (result != hipSuccess)
	{
		printf("cudaMalloc: failed to allocate memory on device for device_input_2\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	result = hipMalloc((void **)&device_output, size);
	if (result != hipSuccess)
	{
		printf("cudaMalloc: failed to allocate memory on device for device_output\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	// copy data from host input buffers to device input buffers
	result = hipMemcpy(device_input_1, host_input_1, size, hipMemcpyHostToDevice);
	if (result != hipSuccess)
	{
		printf("cudaMemcpy: failed to copy data from host buffer host_input_1 to device buffer device_input_1\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	result = hipMemcpy(device_input_2, host_input_2, size, hipMemcpyHostToDevice);
	if (result != hipSuccess)
	{
		printf("cudaMemcpy: failed to copy data from host buffer host_input_2 to device buffer device_buffer_2\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	// configure kernel grid dimensions
	dim3 dimGrid = dim3(element_count, 1, 1);	// 5 blocks in x direction for this grid
	dim3 dimBlock = dim3(1, 1, 1);	// 1 thread per block

	// launch the kernel
	vectAdd_GPU<<< dimGrid, dimBlock >>>(device_input_1, device_input_2, device_output, size);

	// copy data from device output buffer into host output buffer
	result = hipMemcpy(host_output, device_output, size, hipMemcpyDeviceToHost);
	if (result != hipSuccess)
	{
		printf("cudaMemcpy: failed to copy data from device buffer device_output to host buffer host_output\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	// display the output from CUDA compute device
	printf("\n------------------------------------\n");
	printf("Output from the CUDA Compute Device:\n");
	printf("------------------------------------\n");
	for (i = 0; i < element_count; i++)
		printf("%f + %f = %f\n", host_input_1[i], host_input_2[i], host_output[i]);
	printf("\n------------------------------------\n");

	// return all allocated resources
	cleanup();

	return 0;
}

void cleanup(void)
{
	// code
	if (device_output)
	{
		hipFree(device_output);
		device_output = NULL;
	}
	if (device_input_2)
	{
		hipFree(device_input_2);
		device_input_2 = NULL;
	}
	if (device_input_1)
	{
		hipFree(device_input_1);
		device_input_1 = NULL;
	}
	if (host_output)
	{
		free(host_output);
		host_output = NULL;
	}
	if (host_input_2)
	{
		free(host_input_2);
		host_input_2 = NULL;
	}
	if (host_input_1)
	{
		free(host_input_1);
		host_input_1 = NULL;
	}
}
