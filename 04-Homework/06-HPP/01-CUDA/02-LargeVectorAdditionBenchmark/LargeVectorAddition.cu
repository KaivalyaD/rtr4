#include "hip/hip_runtime.h"
/* header files */
// standard headers
#include<stdio.h>

// CUDA headers
#include<hip/hip_runtime.h>
#include"include/helper_timer.h"

/* global variable declarations */
const size_t element_count = 11444777;

float *host_input_1 = NULL;
float *host_input_2 = NULL;
float *host_output = NULL;
float *gold = NULL;

float *device_input_1 = NULL;
float *device_input_2 = NULL;
float *device_output = NULL;

float time_on_CPU = 0.0f;
float time_on_GPU = 0.0f;

/* CUDA kernel */
__global__ void largeVectAdd_GPU(float *a, float *b, float *out, size_t len)
{
	// code
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < len)
	{
		*(out + i) = *(a + i) + *(b + i);
	}
}

/* entry-point function */
int main(void)
{
	// function prototypes
	void populateWithClampedFloatingPointRandomNumbers(float *, size_t);
	void largeVectAdd_CPU(const float *, const float *, float *, const size_t);
	void cleanup(void);

	// variable declarations
	size_t size = element_count * sizeof(float);
	hipError_t result = hipSuccess;

	// code
	// host memory allocation
	host_input_1 = (float *)malloc(size);
	if (!host_input_1)
	{
		printf("malloc: failed to allocate memory for host buffer host_input_1\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	host_input_2 = (float *)malloc(size);
	if (!host_input_2)
	{
		printf("malloc: failed to allocate memory for host buffer host_input_2\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	host_output = (float *)malloc(size);
	if (!host_output)
	{
		printf("malloc: failed to allocate memory for host buffer host_output\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	gold = (float *)malloc(size);
	if (!gold)
	{
		printf("malloc: failed to allocate memory for host buffer gold\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	// populate input host buffers
	populateWithClampedFloatingPointRandomNumbers(host_input_1, element_count);
	populateWithClampedFloatingPointRandomNumbers(host_input_2, element_count);

	// device memory allocation
	result = hipMalloc((void **)&device_input_1, size);
	if (result != hipSuccess)
	{
		printf("hipMalloc: failed to allocate memory for device buffer device_input_1\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	result = hipMalloc((void **)&device_input_2, size);
	if (result != hipSuccess)
	{
		printf("hipMalloc: failed to allocate memory for device buffer device_input_2\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	result = hipMalloc((void **)&device_output, size);
	if (result != hipSuccess)
	{
		printf("hipMalloc: failed to allocate memory for device buffer device_output\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	// copy host input buffers into device input buffers
	result = hipMemcpy(device_input_1, host_input_1, size, hipMemcpyHostToDevice);
	if (result != hipSuccess)
	{
		printf("hipMemcpy: failed to copy host buffer host_input_1 into device buffer device_input_1\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	result = hipMemcpy(device_input_2, host_input_2, size, hipMemcpyHostToDevice);
	if (result != hipSuccess)
	{
		printf("hipMemcpy: failed to copy host buffer host_input_2 into device buffer device_input_2\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	// configuring kernel dimensions
	dim3 dimGrid = dim3((size_t)ceil((float)element_count / 256.0f), 1, 1);
	dim3 dimBlock = dim3(256, 1, 1);

	// setup timer
	StopWatchInterface *timer = NULL;
	sdkCreateTimer(&timer);

	// start timer
	sdkStartTimer(&timer);
	{
		// launch the kernel
		largeVectAdd_GPU<<< dimGrid, dimBlock >>>(device_input_1, device_input_2, device_output, size);
	}
	sdkStopTimer(&timer);
	
	// get time spent on GPU
	time_on_GPU = sdkGetTimerValue(&timer);
	
	// delete timer
	sdkDeleteTimer(&timer);
	timer = NULL;

	// copy device output buffer into host output buffer
	result = hipMemcpy(host_output, device_output, size, hipMemcpyDeviceToHost);
	if (result != hipSuccess)
	{
		printf("hipMemcpy: failed to copy device buffer device_output into host buffer host_output\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	// perform the same addition on host
	// setup timer
	sdkCreateTimer(&timer);

	// start timer
	sdkStartTimer(&timer);
	{
		largeVectAdd_CPU(host_input_1, host_input_2, gold, element_count);
	}
	sdkStopTimer(&timer);

	// get time spent on CPU
	time_on_CPU = sdkGetTimerValue(&timer);

	// delete timer
	sdkDeleteTimer(&timer);
	timer = NULL;

	// measuring GPU accuracy
	const float epsilon = 0.000001;
	int firstInaccurateIndex = -1;
	bool isAccurate = true;

	for (int i = 0; i < element_count; i++)
	{
		float g = gold[i];
		float v = host_output[i];

		if (fabs(g - v) > epsilon)
		{
			isAccurate = false;
			firstInaccurateIndex = i;
			break;
		}
	}

	char statementOfAccuracy[128];
	if (!isAccurate)
	{
		sprintf(statementOfAccuracy, "GPU produced atleast 1 result not within %f of that produced by the CPU at index %d", epsilon, firstInaccurateIndex);
	}
	else
	{
		sprintf(statementOfAccuracy, "All GPU results are within %.6f of CPU results", epsilon);
	}

	char statementOfPerformance[128];
	float timeRatio = time_on_CPU / time_on_GPU;
	if (timeRatio < 1.0f)
	{
		sprintf(statementOfPerformance, "CPU outperformed the GPU by %.6f times", 1.0f / timeRatio);
	}
	else
	{
		sprintf(statementOfPerformance, "GPU outperformed the CPU by %.6f times", timeRatio);
	}

	// display output of the benchmark
	printf("\n");
	printf("+------------------------------------------------------------------------------+\n");
	printf("+                      CPU vs GPU on Large Vector Addition                     +\n");
	printf("+                                                                              +\n");
	printf("+                                                                              +\n");
	printf("+ vectorA begins at 0 with %.6f and ends at %lld with %.6f         +\n", host_input_1[0], element_count - 1, host_input_1[element_count - 1]);
	printf("+ vectorB begins at 0 with %.6f and ends at %lld with %.6f         +\n", host_input_2[0], element_count - 1, host_input_2[element_count - 1]);
	printf("+                                                                              +\n");
	printf("+ on CPU:                                                                      +\n");
	printf("+ \tvectorOut begins at 0 with %.6f and ends at %lld with %.6f +\n", gold[0], element_count - 1, gold[element_count - 1]);
	printf("+                                                                              +\n");
	printf("+ \ttime taken on the CPU: %.6f ms                                    +\n", time_on_CPU);
	printf("+                                                                              +\n");
	printf("+------------------------------------------------------------------------------+\n");
	printf("+                                                                              +\n");
	printf("+ on GPU:                                                                      +\n");
	printf("+ \tvectorOut begins at 0 with %.6f and ends at %lld with %.6f +\n", host_output[0], element_count - 1, host_output[element_count - 1]);
	printf("+                                                                              +\n");
	printf("+ \tCUDA kernel grid dimensions: %d x %d x %d                             +\n", dimGrid.x, dimGrid.y, dimGrid.z);
	printf("+ \tCUDA kernel block dimensions: %d x %d x %d                              +\n", dimBlock.x, dimBlock.y, dimBlock.z);
	printf("+                                                                              +\n");
	printf("+ \ttime taken on the GPU: %.6f ms                                     +\n", time_on_GPU);
	printf("+                                                                              +\n");
	printf("+ Comparison of CPU and GPU Accuracy in Large Vector Addition:                 +\n");
	printf("+ %s                           +\n", statementOfAccuracy);
	printf("+                                                                              +\n");
	printf("+ Comparison of CPU and GPU Performance in Large Vector Addition:              +\n");
	printf("+ %s                                 +\n", statementOfPerformance);
	printf("+------------------------------------------------------------------------------+\n");

	// cleanup
	cleanup();

	return 0;
}

void populateWithClampedFloatingPointRandomNumbers(float *v, size_t len)
{
	// code
	const float scalar = 1.0f / (float)RAND_MAX;

	for (int i = 0; i < len; i++)
	{
		*(v + i) = scalar * rand();
	}
}

void largeVectAdd_CPU(const float *a, const float *b, float *out, const size_t len)
{
	// code
	for (int i = 0; i < len; i++)
	{
		*(out + i) = *(a + i) + *(b + i);
	}
}

void cleanup(void)
{
	// code
	if (device_output)
	{
		hipFree(device_output);
		device_output = NULL;
	}
	if (device_input_2)
	{
		hipFree(device_input_2);
		device_input_2 = NULL;
	}
	if (device_input_1)
	{
		hipFree(device_input_1);
		device_input_1 = NULL;
	}
	if (gold)
	{
		free(gold);
		gold = NULL;
	}
	if (host_output)
	{
		free(host_output);
		host_output = NULL;
	}
	if (host_input_2)
	{
		free(host_input_2);
		host_input_2 = NULL;
	}
	if (host_input_1)
	{
		free(host_input_1);
		host_input_1 = NULL;
	}
}
