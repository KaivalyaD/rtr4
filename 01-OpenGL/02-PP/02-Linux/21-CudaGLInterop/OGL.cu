#include "hip/hip_runtime.h"
/* standard headers */
#include <stdio.h>  // for standard I/O
#include <stdlib.h> // for exit()
#include <memory.h> // for memset()

/* vector math */
#include "vmath.h"

/* X11 headers */
#include <X11/Xlib.h>
#include <X11/Xutil.h>  // for XVisualInfo
#include <X11/XKBlib.h> // for the keyboard

/* GLEW headers */
#include <GL/glew.h>

/* OpenGL headers */
#include <GL/gl.h>  // the OpenGL API
#include <GL/glx.h> // API for GL/XServer bridging

/* Cuda-GL interop headers */
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

/* macros */
#define WIN_WIDTH 1200
#define WIN_HEIGHT 800

#define MAX_MESH_WIDTH 2048
#define MIN_MESH_WIDTH 128
#define MAX_MESH_HEIGHT 2048
#define MIN_MESH_HEIGHT 128

/* enums */
enum {
    KVD_ATTRIBUTE_POSITION = 0,
    KVD_ATTRIBUTE_COLOR,
    KVD_ATTRIBUTE_NORMAL,
    KVD_ATTRIBUTE_TEXTURE0,
};

/* typedefs */
typedef GLXContext (*glXCreateContextAttribsARBProc)(Display *, GLXFBConfig, GLXContext, Bool, const int*);

/* global variables */
/* windowing */
Display *display = NULL;
XVisualInfo *visualInfo = NULL;
Colormap colormap;
Window window;
Bool fullScreen = False;
Bool activeWindow = False;
FILE *logFile = NULL;

/* glX variables */
glXCreateContextAttribsARBProc glXCreateContextAttribsARB = NULL;
GLXFBConfig glxFBConfig;
GLXContext glxContext = NULL;

/* related to the OpenGL programmable pipeline */
GLuint shaderProgramObject;

/* globals for rendering */
GLuint vao;
GLuint vbo;
GLuint gpuVbo;
GLuint mvpMatrixUniform;

vmath::mat4 perspectiveProjectionMatrix;

unsigned int meshWidth = MAX_MESH_WIDTH;
unsigned int meshHeight = MAX_MESH_HEIGHT;
unsigned int arraySize = meshWidth * meshHeight * 4;

float positions[MAX_MESH_HEIGHT][MAX_MESH_WIDTH][4];
float animTime = 0.0f;
bool onGPU = false;

hipGraphicsResource_t graphicsResource = NULL;

/* cuda kernel to animate a wave on the point mesh */
__global__ void animateSineWaveOnGPU(float4 *positions, unsigned int width, unsigned int height, float phase)
{
    float freq = 4.0f;
    unsigned int z = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

    if(x < width && z < height)
    {
        float u = (float)x / (float)width;
        float v = (float)z / (float)height;

        u = (2.0f * u) - 1.0f;
        v = (2.0f * v) - 1.0f;

        positions[(x * width) + z] = make_float4(u, 0.5f * sinf((u * freq) + phase) * cosf((v * freq) + phase), v, 1.0f);
    }
}

/* entry-point function */
int main(int argc, char *argv[])
{
    // function declarations
    void toggleFullscreen(void);
    int initialize(void);
    void resize(int, int);
    void update(void);
    void render(void);
    void uninitialize(void);

    // local variables
    int defaultScreen, defaultDepth;
    GLXFBConfig *glxFBConfigs = NULL;
    GLXFBConfig bestGLXFBConfig;
    XVisualInfo *tempXVisualInfo = NULL;
    int cntFBConfigs;
    XSetWindowAttributes windowAttributes;
    int styleMask = 0;
    Atom wm_delete_window_atom;
    XEvent event;
    KeySym keysym;
    int screenWidth, screenHeight;
    char keys[26];
    
    int framebufferAttributes[] = {
        GLX_X_RENDERABLE, True,              // hardware accelerated renderable
        GLX_DRAWABLE_TYPE, GLX_WINDOW_BIT,   // draw to a window
        GLX_RENDER_TYPE, GLX_RGBA_BIT,       // the render type is an RGBA frame
        GLX_X_VISUAL_TYPE, GLX_TRUE_COLOR,
        GLX_RED_SIZE, 8,
        GLX_GREEN_SIZE, 8,
        GLX_BLUE_SIZE, 8,
        GLX_ALPHA_SIZE, 8,
        GLX_STENCIL_SIZE, 8,
        GLX_DEPTH_SIZE, 24,
        GLX_DOUBLEBUFFER, True,
        None
    };
    Bool done = False;
    int ret;
    int winWidth, winHeight;

    // code
    logFile = fopen("log.txt", "w");
    if(!logFile)
    {
        fprintf(stderr, "fopen: failed to open log.txt\n");
        uninitialize();
        exit(EXIT_FAILURE);
    }
    fprintf(logFile, "successfully created log file\n");

    display = XOpenDisplay(NULL);
    if(!display)
    {
        fprintf(stderr, "error: XOpenDisplay() failed\n");
        uninitialize();
        exit(EXIT_FAILURE);
    }

    defaultScreen = XDefaultScreen(display);
    defaultDepth = XDefaultDepth(display, defaultScreen);
    defaultDepth;

    // getting all conforming FB configurations
    glxFBConfigs = glXChooseFBConfig(display, defaultScreen, framebufferAttributes, &cntFBConfigs);
    if(!glxFBConfigs)
    {
        fprintf(logFile, "error: glXChooseFBConfig() failed\n");
        uninitialize();
        exit(EXIT_FAILURE);
    }
    fprintf(logFile, "found %d conforming framebuffer configurations:\n", cntFBConfigs);

    // choosing the best FB configuration but getting both -the best and the worst
    int bestFramebufferConfigIdx = -1;
    int worstFramebufferConfigIdx = -1;
    int bestCountOfSamples = -1;
    int worstCountOfSamples = 999;

    for(int i = 0; i < cntFBConfigs; i++)
    {
        tempXVisualInfo = glXGetVisualFromFBConfig(display, glxFBConfigs[i]);
        if(!tempXVisualInfo)
            continue;       // if an FB config has no XVisualInfo, it cannot be the best
        
        int samples, sampleBuffers;
        glXGetFBConfigAttrib(display, glxFBConfigs[i], GLX_SAMPLE_BUFFERS, &sampleBuffers);
        glXGetFBConfigAttrib(display, glxFBConfigs[i], GLX_SAMPLES, &samples);

        fprintf(logFile, "[%d] XVisualInfo (visualid 0x%lx) has: %d sample buffers and %d samples\n", (i + 1), tempXVisualInfo->visualid, sampleBuffers, samples);

        if((bestFramebufferConfigIdx < 0) || (sampleBuffers && (samples > bestCountOfSamples)))
        {
            bestFramebufferConfigIdx = i;
            bestCountOfSamples = samples;
        }

        if((worstFramebufferConfigIdx >= 999) || !sampleBuffers || (samples < worstCountOfSamples))
        {
            // this may be not the worst of the worst, but only as good as unusable
            worstFramebufferConfigIdx = i;
            worstCountOfSamples = samples;
        }

        XFree(tempXVisualInfo);
        tempXVisualInfo = NULL;
    }
    bestGLXFBConfig = glxFBConfigs[bestFramebufferConfigIdx];
    glxFBConfig = bestGLXFBConfig;
    
    XFree(glxFBConfigs);
    glxFBConfigs = NULL;

    // obtaining visual info from the best FB configuration
    visualInfo = glXGetVisualFromFBConfig(display, bestGLXFBConfig);
    fprintf(logFile, "visualid of the best XVisualInfo found: 0x%lx\n", visualInfo->visualid);

    memset(&windowAttributes, 0, sizeof(XSetWindowAttributes));
    windowAttributes.border_pixel = 0;
    windowAttributes.background_pixel = XBlackPixel(display, defaultScreen);
    windowAttributes.background_pixmap = 0;
    windowAttributes.colormap = XCreateColormap(display, RootWindow(display, visualInfo->screen), visualInfo->visual, AllocNone);
    windowAttributes.event_mask |= ExposureMask | KeyPressMask | StructureNotifyMask | FocusChangeMask;

    colormap = windowAttributes.colormap;

    styleMask |= CWBorderPixel | CWBackPixel | CWColormap | CWEventMask;

    window = XCreateWindow(
        display,
        RootWindow(display, visualInfo->screen),
        0,
        0,
        WIN_WIDTH,
        WIN_HEIGHT,
        0,
        visualInfo->depth,
        InputOutput,
        visualInfo->visual,
        styleMask,
        &windowAttributes
    );
    if(!window)
    {
        fprintf(stderr, "error: XCreateWindow() failed\n");
        uninitialize();
        exit(EXIT_FAILURE);
    }

    XStoreName(display, window, "Kaivalya Vishwakumar Deshpande: OpenGL");

    wm_delete_window_atom = XInternAtom(display, "WM_DELETE_WINDOW", True); // True: create the atom in all cases: whether it exists before or not
    XSetWMProtocols(display, window, &wm_delete_window_atom, 1);

    XMapWindow(display, window);

    // centering the window
    screenWidth = XWidthOfScreen(XScreenOfDisplay(display, defaultScreen));
    screenHeight = XHeightOfScreen(XScreenOfDisplay(display, defaultScreen));
    XMoveWindow(display, window, ((screenWidth - WIN_WIDTH) / 2), ((screenHeight - WIN_HEIGHT) / 2));

    // initialize OpenGL
    ret = initialize();
    if(ret == -1)
    {
        fprintf(logFile, "error: glXGetProcAddressARB() failed\n");
        uninitialize();
        exit(EXIT_FAILURE);
    }
    fprintf(logFile, "OpenGL initialized successfully\n");

    // game loop
    while(!done)
    {
        while(XPending(display))
        {
            XNextEvent(display, &event);
            switch(event.type)
            {
            case MapNotify:
                break;

            case KeyPress:
                keysym = XkbKeycodeToKeysym(display, event.xkey.keycode, 0, 0);
                switch(keysym)
                {
                case XK_Escape:
                    done = True;
                    break;

                case XK_space:
                    onGPU = !onGPU;
                    break;

                default:
                    break;
                }

                XLookupString(&event.xkey, keys, sizeof(keys), NULL, NULL);
                switch(*keys)
                {
                case 'F':
                case 'f':
                    toggleFullscreen();
                    break;

                default:
                    break;
                }
                break;
            
            case FocusIn:   /* WM_SETFOCUS in Win32 */
                activeWindow = True;
                break;

            case FocusOut:  /* WM_KILLFOCUS in Win32 */
                activeWindow = False;
                break;

            case ConfigureNotify:
                winWidth = event.xconfigure.width;
                winHeight = event.xconfigure.height;
                resize(winWidth, winHeight);
                break;

            case 33:
                done = True;
                break;

            default:
                break;
            }
        }
        
        if(activeWindow)
        {
            // update the scene
            update();

            // render the scene
            render();
        }
    }
    uninitialize();

    return 0;
}

void toggleFullscreen(void)
{
    // local variables
    Atom wm_current_state_atom;
    Atom wm_fullscreen_state_atom;
    XEvent event;

    // code
    if(fullScreen)
        fullScreen = False;
    else
        fullScreen = True;

    wm_current_state_atom = XInternAtom(display, "_NET_WM_STATE", False);
    wm_fullscreen_state_atom = XInternAtom(display, "_NET_WM_STATE_FULLSCREEN", False);

    memset(&event, 0, sizeof(XEvent));
    event.type = ClientMessage;
    event.xclient.window = window;
    event.xclient.message_type = wm_current_state_atom;
    event.xclient.format = 32;
    event.xclient.data.l[0] = fullScreen ? 1 : 0;
    event.xclient.data.l[1] = wm_fullscreen_state_atom;

    XSendEvent(display, RootWindow(display, visualInfo->screen), False, SubstructureNotifyMask, &event);
}

int initialize(void)
{
    // function prototypes
    void logGLInfo(void);
    void uninitialize(void);
    void resize(int, int);
    void checkCudaError(const char *, int);

    // variable declarations
    GLint contextAttributes[] = {
        GLX_CONTEXT_MAJOR_VERSION_ARB, 4,   // for an OpenGL 4.6 context
        GLX_CONTEXT_MINOR_VERSION_ARB, 6,
        GLX_CONTEXT_PROFILE_MASK_ARB, GLX_CONTEXT_CORE_PROFILE_BIT_ARB, // core-profile OpenGL
        None
    };

    // code
    glXCreateContextAttribsARB = (glXCreateContextAttribsARBProc)glXGetProcAddressARB((GLubyte *)"glXCreateContextAttribsARB");
    if(!glXCreateContextAttribsARB)
        return -1;

    // passing the 2nd last argument as True means enabling hardware support for rendering
    glxContext = glXCreateContextAttribsARB(display, glxFBConfig, NULL, True, contextAttributes);
    if(!glxContext)
    {
        GLint contextAttributesFallback[] = {
            GLX_CONTEXT_MAJOR_VERSION_ARB, 1,   // this does not mean you get OpenGL 1.0 strictly, but the highest supported version
            GLX_CONTEXT_MINOR_VERSION_ARB, 0,
            None
        };

        glxContext = glXCreateContextAttribsARB(display, glxFBConfig, NULL, True, contextAttributesFallback);
        fprintf(logFile, "cannot support OpenGL version 4.6; falling back to the default\n");
    }
    else
        fprintf(logFile, "OpenGL 4.6 is fully supported\n");

    // checking whether hardware/direct rendering is supported or not
    if(glXIsDirect(display, glxContext))
        fprintf(logFile, "hardware rendering is supported\n");
    else
        fprintf(logFile, "hardware rendering is not supported\n");

    glXMakeCurrent(display, window, glxContext);

    // glew initialization
	if (glewInit() != GLEW_OK)
		return -5;

    // cuda initialization
    int devCount = 0;
    hipGetDeviceCount(&devCount); checkCudaError("hipGetDeviceCount", __LINE__);
    if(devCount <= 0)
    {
        fprintf(logFile, "fatal error: initialize(): no cuda supporting devices with compute capability >= 2.0 found on this machine");
        uninitialize();
        exit(EXIT_FAILURE);
    }
    hipSetDevice(0); checkCudaError("hipSetDevice", __LINE__);  // select 0th device as default

	// logging OpenGL info
	logGLInfo();

	// vertex shader
	int status = 0;
	int infoLogLength = 0;
	char *infoLog = NULL;

	const GLchar *vertexShaderSourceCode =
		"#version 460 core\n" \
        "\n" \
        "in vec4 a_position;\n" \
        "\n" \
        "uniform mat4 u_mvpMatrix;\n" \
        "\n" \
		"void main(void)\n" \
		"{\n" \
			"gl_Position = u_mvpMatrix * a_position;\n" \
		"}\n";

	GLuint vertexShaderObject = glCreateShader(GL_VERTEX_SHADER);
	glShaderSource(vertexShaderObject, 1, (const GLchar **)&vertexShaderSourceCode, NULL);
	glCompileShader(vertexShaderObject);
	glGetShaderiv(vertexShaderObject, GL_COMPILE_STATUS, &status);
	if (status == GL_FALSE)
	{
		fprintf(logFile, "*** vertex shader compilation errors ***\n");

		glGetShaderiv(vertexShaderObject, GL_INFO_LOG_LENGTH, &infoLogLength);
		if (infoLogLength > 0)
		{
			infoLog = (char *)malloc(infoLogLength * sizeof(char));
			if (infoLog)
			{
				GLsizei written = 0;

				glGetShaderInfoLog(vertexShaderObject, infoLogLength * sizeof(char), &written, infoLog);
				fprintf(logFile, "vertex shader compilation log (%d bytes):\n%s\n", written, infoLog);
				free(infoLog);
				infoLog = NULL;
			}
			else
				fprintf(logFile, "\tmalloc: cannot allocate memory to hold the compilation log\n");
		}
		else
			fprintf(logFile, "\tthere is nothing to print\n");

		uninitialize();
        exit(EXIT_FAILURE);
	}
	fprintf(logFile, "vertex shader was compiled without errors\n");

	// fragment shader
	status = 0;
	infoLogLength = 0;
	infoLog = NULL;

	const GLchar *fragmentShaderSourceCode =
		"#version 460 core\n" \
        "\n" \
        "out vec4 FragColor;\n" \
        "\n" \
		"void main(void)\n" \
		"{\n" \
			"FragColor = vec4(1.0, 0.5, 0.0, 1.0);\n" \
		"}\n";

	GLuint fragmentShaderObject = glCreateShader(GL_FRAGMENT_SHADER);
	glShaderSource(fragmentShaderObject, 1, (const GLchar **)&fragmentShaderSourceCode, NULL);
	glCompileShader(fragmentShaderObject);
	glGetShaderiv(fragmentShaderObject, GL_COMPILE_STATUS, &status);
	if (status == GL_FALSE)
	{
		fprintf(logFile, "*** fragment shader compilation errors ***\n");

		glGetShaderiv(fragmentShaderObject, GL_INFO_LOG_LENGTH, &infoLogLength);
		if (infoLogLength > 0)
		{
			infoLog = (char *)malloc(infoLogLength * sizeof(char));
			if (infoLog)
			{
				GLsizei written = 0;

				glGetShaderInfoLog(fragmentShaderObject, infoLogLength, &written, infoLog);
				fprintf(logFile, "fragment shader compilation log (%d bytes):\n%s\n", written, infoLog);
				free(infoLog);
				infoLog = NULL;
			}
			else
				fprintf(logFile, "\tmalloc: cannot allocate memory to hold the compilation log\n");
		}
		else
			fprintf(logFile, "\tthere is nothing to print\n");

		uninitialize();
        exit(EXIT_FAILURE);
	}
	fprintf(logFile, "fragment shader was compiled without errors\n");

	// shader program object
	status = 0;
	infoLogLength = 0;
	infoLog = NULL;

	shaderProgramObject = glCreateProgram();
	glAttachShader(shaderProgramObject, vertexShaderObject);
	glAttachShader(shaderProgramObject, fragmentShaderObject);
    glBindAttribLocation(shaderProgramObject, KVD_ATTRIBUTE_POSITION, "a_position");
	glLinkProgram(shaderProgramObject);
	glGetProgramiv(shaderProgramObject, GL_LINK_STATUS, &status);
	if (status == GL_FALSE)
	{
		fprintf(logFile, "*** there were linking errors ***\n");

		glGetProgramiv(shaderProgramObject, GL_INFO_LOG_LENGTH, &infoLogLength);
		if (infoLogLength > 0)
		{
			infoLog = (char *)malloc(infoLogLength * sizeof(char));
			if (infoLog)
			{
				GLsizei written = 0;

				glGetProgramInfoLog(shaderProgramObject, infoLogLength * sizeof(char), &written, infoLog);
				fprintf(logFile, "\tlink time info log (%d bytes):\n%s\n", written, infoLog);
				free(infoLog);
				infoLog = NULL;
			}
			else
				fprintf(logFile, "\tmalloc: cannot allocate memory to hold the linking log\n");
		}
		else
			fprintf(logFile, "\tthere is nothing to print\n");

		uninitialize();
        exit(EXIT_FAILURE);
	}
	fprintf(logFile, "shader program was linked without errors\n");

    /* post link processing */
    mvpMatrixUniform = glGetUniformLocation(shaderProgramObject, "u_mvpMatrix");

    glGenVertexArrays(1, &vao);
    glBindVertexArray(vao);
    {
        // buffer for cpu-computed sine wave
        glGenBuffers(1, &vbo);
        glBindBuffer(GL_ARRAY_BUFFER, vbo);
            glBufferData(GL_ARRAY_BUFFER, arraySize * sizeof(float), NULL, GL_DYNAMIC_DRAW);
        glBindBuffer(GL_ARRAY_BUFFER, 0);

        // buffer for gpu-computed sine wave
        glGenBuffers(1, &gpuVbo);
        glBindBuffer(GL_ARRAY_BUFFER, gpuVbo);
            glBufferData(GL_ARRAY_BUFFER, arraySize * sizeof(float), NULL, GL_DYNAMIC_DRAW);
        glBindBuffer(GL_ARRAY_BUFFER, 0);
    }
    glBindVertexArray(0);

    // bring vertex positions array into a known state
    for(unsigned int i = 0; i < meshHeight; i++)
    {
        for(unsigned int j = 0; j < meshWidth; j++)
        {
            positions[i][j][0] = 0.0f;
            positions[i][j][1] = 0.0f;
            positions[i][j][2] = 0.0f;
            positions[i][j][3] = 0.0f;
        }
    }

    // create cuda/opengl interop resource
    // register gpuVbo as a writable graphics resource to cuda; discard memory after being used
    hipGraphicsGLRegisterBuffer(&graphicsResource, gpuVbo, cudaGraphicsMapFlagsWriteDiscard); checkCudaError("hipGraphicsGLRegisterBuffer", __LINE__);
    
    // setting up depth
    glClearDepth(1.0f);
    glEnable(GL_DEPTH_TEST);
    glDepthFunc(GL_LEQUAL);

    // setting the clear color
    glClearColor(0.0f, 0.0f, 0.0f, 1.0f);

    // initializing the perspective projection matrix
    perspectiveProjectionMatrix = vmath::mat4::identity();

    // warm-up resize
    resize(WIN_WIDTH, WIN_HEIGHT);

    return 0;
}

void logGLInfo(void)
{
	// variable declarations
	GLint numExtensions = 0;

	// code
	fprintf(logFile, "\n-------------------- OpenGL Properties --------------------\n\n");
	fprintf(logFile, "OpenGL Vendor   : %s\n", glGetString(GL_VENDOR));
	fprintf(logFile, "OpenGL Renderer : %s\n", glGetString(GL_RENDERER));
	fprintf(logFile, "OpenGL Version  : %s\n", glGetString(GL_VERSION));
	fprintf(logFile, "GLSL Version    : %s\n", glGetString(GL_SHADING_LANGUAGE_VERSION));

	fprintf(logFile, "\n-------------------- OpenGL Extensions --------------------\n\n");
	glGetIntegerv(GL_NUM_EXTENSIONS, &numExtensions);
	fprintf(logFile, "Number of supported extensions : %d\n\n", numExtensions);
	for (int i = 0; i < numExtensions; i++)
	{
		fprintf(logFile, "%s\n", glGetStringi(GL_EXTENSIONS, i));
	}
	fprintf(logFile, "-------------------------------------------------------------\n\n");
}

void resize(int width, int height)
{
    // variable declarations
    GLfloat aspectRatio;

    // code
    if(height == 0)
        height = 1;
    
    glViewport(0, 0, (GLsizei)width, (GLsizei)height);

    aspectRatio = (GLfloat)width / (GLfloat)height;
    perspectiveProjectionMatrix = vmath::perspective(
        45.0f,
        aspectRatio,
        0.1f,
        100.0f
    );
}

void render(void)
{
    // function prototypes
    void animateSineWaveOnCPU(unsigned int, unsigned int, float);
    void checkCudaError(const char *, int);

    // variable declarations
    vmath::mat4 modelViewMatrix;
    vmath::mat4 modelViewProjectionMatrix;
    float4 *pPositions = NULL;
    size_t numMappedBytes = 0U;

    static char title[255];

    // code
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	glUseProgram(shaderProgramObject);
    {
        // transformations
        modelViewMatrix = vmath::translate(0.0f, 0.0f, -1.5f);
        modelViewProjectionMatrix = perspectiveProjectionMatrix * modelViewMatrix;
        glUniformMatrix4fv(mvpMatrixUniform, 1, GL_FALSE, modelViewProjectionMatrix);

        glBindVertexArray(vao);
        {
            if(!onGPU)
            {
                sprintf(title, "Kaivalya Deshpande: OpenGL-CUDA | Points Rendered = %d | Running on CPU", meshWidth * meshHeight);
                XStoreName(display, window, title);

                animateSineWaveOnCPU(meshWidth, meshHeight, animTime);
                glBindBuffer(GL_ARRAY_BUFFER, vbo);
                    glBufferData(GL_ARRAY_BUFFER, arraySize * sizeof(float), positions, GL_DYNAMIC_DRAW);
                    glVertexAttribPointer(KVD_ATTRIBUTE_POSITION, 4, GL_FLOAT, GL_FALSE, 0, NULL);
                    glEnableVertexAttribArray(KVD_ATTRIBUTE_POSITION);
                glBindBuffer(GL_ARRAY_BUFFER, 0);
            }
            else
            {
                sprintf(title, "Kaivalya Deshpande: OpenGL-CUDA | Points Rendered = %d | Running on GPU", meshWidth * meshHeight);
                XStoreName(display, window, title);

                // map cuda graphics resource
                hipGraphicsMapResources(1, &graphicsResource, 0); checkCudaError("hipGraphicsMapResources", __LINE__);

                // get pointer to the mapped resource
                hipGraphicsResourceGetMappedPointer((void **)&pPositions, &numMappedBytes, graphicsResource); checkCudaError("hipGraphicsResourceGetMappedPointer", __LINE__);

                // configure and launch the kernel
                dim3 block(8, 8, 1);
                dim3 grid((meshWidth / block.x) + 1, (meshHeight / block.y) + 1, 1);
                animateSineWaveOnGPU<<< grid, block >>>(pPositions, meshWidth, meshHeight, animTime); checkCudaError("(launcher) animateSineWaveOnGPU<<< >>>", __LINE__);

                // unmap graphics resource
                hipGraphicsUnmapResources(1, &graphicsResource, 0); checkCudaError("hipGraphicsUnmapResources", __LINE__);
            
                // perform drawing rituals as usual
                glBindBuffer(GL_ARRAY_BUFFER, gpuVbo);
                    glVertexAttribPointer(KVD_ATTRIBUTE_POSITION, 4, GL_FLOAT, GL_FALSE, 0, NULL);
                    glEnableVertexAttribArray(KVD_ATTRIBUTE_POSITION);
                glBindBuffer(GL_ARRAY_BUFFER, 0);
            }

            glDrawArrays(GL_POINTS, 0, meshWidth * meshHeight);
        }
        glBindVertexArray(0);
    }
	glUseProgram(0);

    glXSwapBuffers(display, window);
}

void update(void)
{
    // code
    animTime += 0.01f;
    if(animTime > 360.0f)
        animTime -= 360.0f;
}

void uninitialize(void)
{
    // local variables
    GLXContext currentContext;

    // code
    if(fullScreen)
    {
        toggleFullscreen();
    }

    if(gpuVbo)
    {
        if(graphicsResource)
        {
            hipGraphicsUnregisterResource(graphicsResource);
            graphicsResource = NULL;
        }
        glDeleteBuffers(1, &gpuVbo);
        gpuVbo = 0U;
    }

    if(vbo)
    {
        glDeleteBuffers(1, &vbo);
        vbo = 0U;
    }

    if(vao)
    {
        glDeleteVertexArrays(1, &vao);
        vao = 0U;
    }

    if (shaderProgramObject)
	{
		GLsizei numAttachedShaders = 0;
		GLuint *shaderObjects = NULL;

		glUseProgram(shaderProgramObject);
		
		glGetProgramiv(shaderProgramObject, GL_ATTACHED_SHADERS, &numAttachedShaders);
		shaderObjects = (GLuint *)malloc(numAttachedShaders * sizeof(GLuint));
		glGetAttachedShaders(shaderProgramObject, numAttachedShaders, &numAttachedShaders, shaderObjects);

		for (GLsizei i = 0; i < numAttachedShaders; i++)
		{
			glDetachShader(shaderProgramObject, shaderObjects[i]);
			glDeleteShader(shaderObjects[i]);
			shaderObjects[i] = 0;
		}
		free(shaderObjects);
		shaderObjects = NULL;
		fprintf(logFile, "detached and deleted %d shader objects\n", numAttachedShaders);

		glUseProgram(0);
		glDeleteProgram(shaderProgramObject);
		shaderProgramObject = 0;
		fprintf(logFile, "deleted shader program object\n");
	}

    currentContext = glXGetCurrentContext();
    if(currentContext && currentContext == glxContext)
    {
        glXMakeCurrent(display, 0, 0);
        currentContext = NULL;
    }

    if(glxContext)
    {
        glXDestroyContext(display, glxContext);
        glxContext = NULL;
    }

    if(visualInfo)
    {
        free(visualInfo);
        visualInfo = NULL;
    }

    if(window)
    {
        XDestroyWindow(display, window);
        window = 0;
    }

    if(colormap)
    {
        XFreeColormap(display, colormap);
        colormap = 0;
    }

    if(display)
    {
        XCloseDisplay(display);
        display = NULL;
    }

    if(logFile)
    {
        fprintf(logFile, "closing log file\n");
        fclose(logFile);
        logFile = NULL;
    }
}

void animateSineWaveOnCPU(unsigned int width, unsigned int height, float phase)
{
    // local variables
    float u, v;
    float freq = 4.0f;

    // code
    for(unsigned int i = 0; i < height; i++)
    {
        for(unsigned int j = 0; j < width; j++)
        {
            // u, v -> [0, 1]
            u = (float)j / (float)width;
            v = (float)i / (float)height;

            // u, v -> [-1, 1]
            u = (2.0f * u) - 1.0f;
            v = (2.0f * v) - 1.0f;

            positions[i][j][0] = u;
            positions[i][j][1] = 0.5f * sinf((u * freq) + phase) * cosf((v * freq) + phase);
            positions[i][j][2] = v;
            positions[i][j][3] = 1.0f;
        }
    }
}

/* cuda error handling utility function */
void checkCudaError(const char *func, int lineNum)
{
    // code
    hipError_t code = hipGetLastError();
    if(code != hipSuccess)
    {
        fprintf(logFile, "cuda error: %f() : line %d : %s(%d)\n", func, lineNum, hipGetErrorString(code), code);
        uninitialize();
        exit(EXIT_FAILURE);
    }
}